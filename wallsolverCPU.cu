/*	CPU Based Wallsolver 

	nvcc wallsolverCPU.cu -o testCPU

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <math.h>


#define SPACE_LENGTH 5		// Spaces Size of rows / columns 
#define SPACE_WIDTH 5 

#define WALL_LENGTH 4		// Walls size of rows/colums
#define WALL_WIDTH 4	


typedef enum wall {
	UP, DOWN, LEFT, RIGHT
} wall;


typedef struct space {
	bool up, down, left, right, start, finish;

} space;



bool checkWallCollisions(wall *walls, int idx) {
	/* 	Make sure no walls overlap
		For each wall, identify the neighboring walls if they exist
		determine if a neighbor caused a conflict
		Return TRUE if there is a collision
	*/
	int i = idx / WALL_LENGTH;
	int j = idx % WALL_WIDTH;

	bool colUp = false;
	bool colDown = false;
	bool colLeft = false;
	bool colRight = false;

	wall up, down, left, right;

	if (j < 4) {
		right = walls[idx + 1];
		colRight = (walls[idx] == RIGHT) && (right == LEFT);
	}

	if (j > 0) {
		left = walls[idx - 1];
		colLeft = (walls[idx] == LEFT) && (left == RIGHT);
	}

	if (i < 4) {
		down = walls[idx + WALL_WIDTH];
		colDown = (walls[idx] == DOWN) && (down == UP);
	} 

	if (i > 0) {
		up = walls[idx - WALL_LENGTH];
		colUp = (walls[idx] == UP) && (up == DOWN);
	}

	// Returns true if there is a collision
	return (colUp || colDown || colLeft || colRight);
}


void generateBoard(space *board, wall *walls) {
	/* 	Generate the board
		For each wall, identify the board spaces that it effects
		Determine the effect of each affected space's mobility
	*/
	int numSpaces = WALL_LENGTH * WALL_WIDTH;

	for (int i = 0; i < WALL_WIDTH; i++) {

		for (int j = 0; j < WALL_LENGTH; j++) {
			int idx = (i * WALL_LENGTH) + j;

			printf("Maze Generated: %d - %d\n", idx, walls[idx]);

			// Determine the 4 adjacent spaces to this wall
			int TL = idx + i;
			int TR = TL +1;
			int BL = TL + SPACE_LENGTH;
			int BR = BL +1;

			if (board[TL].right) board[TL].right = (walls[idx] != UP);
			if (board[TL].down) board[TL].down = (walls[idx] != LEFT);

			if (board[TR].left) board[TR].left = board[TL].right;
			if (board[TR].down) board[TR].down = (walls[idx] != RIGHT);

			if (board[BL].right) board[BL].right = (walls[idx] != DOWN);
			if (board[BL].up) board[BL].up = board[TL].down;

			if (board[BR].left) board[BR].left = board[BL].right;
			if (board[BR].up) board[BR].up = board[TR].down;

		}

	}

	board[0].start = true;
	board[numSpaces - 1].finish = true;

}


void boardInit(space *board) {
	// Initialize the board, blank
	for (int i = 0; i < SPACE_LENGTH; i++) {

	for (int j = 0; j < SPACE_WIDTH; j++) {
			int idx = (i * SPACE_WIDTH) + j;
			//board[idx] = blankSpace;

			/*
			if (i == 0) board[idx].up = false;
			if (j == 0) board[idx].left = false;
			if (i == (SPACE_WIDTH - 1)) board[idx].down = false;
			if (j == (SPACE_LENGTH - 1)) board[idx].right = false;
			*/

			// Better to avoid divergence
			board[idx].up = (i != 0);
			board[idx].left = (j != 0);
			board[idx].down = (i != (SPACE_WIDTH - 1));
			board[idx].right = (j != (SPACE_LENGTH - 1));
			board[idx].start = false;
			board[idx].finish = false;
		}

	}
}


void generateWalls(wall *walls) {
	/*	Randomly generate the walls for the board

	*/
	srand(1024);
	for (int i = 0; i < WALL_WIDTH; i++) {

		for (int j = 0; j < WALL_LENGTH; j++) {
			int idx = (i * WALL_LENGTH) + j; 	// == walls[i][j];

			walls[idx] = (wall)(rand() % 4);
			
			printf("IDX %d - %d\n", idx, walls[idx]);
		}

	}

	// Check for any wall collisions and re-randomize if necessary

	for (int i = 0; i < WALL_LENGTH; i++) {

		for (int j = 0; j < WALL_WIDTH; j++) {
			int idx = (i * WALL_WIDTH) + j;

			while (checkWallCollisions(walls, idx)) {
				printf("IDX No Overlap: %d - %d\n", idx, walls[idx]);
				walls[idx] = (wall)(rand() % 4);			
			}
		}
	}

}


void outputBoard(space *in) {

	for (int i = 0; i < SPACE_WIDTH; i++) {

		for (int j = 0; j < SPACE_LENGTH; j++) {
			int idx = (i * SPACE_WIDTH) + j;	// == board[i][j];

			printf("Space #: %d, UP: %d, DOWN: %d, LEFT: %d, RIGHT: %d \n", idx, in[idx].up, in[idx].down, in[idx].left, in[idx].right);

		}


	}

}


int main(int argc, char const *argv[])
{
	
	int numSpaces = SPACE_LENGTH * SPACE_WIDTH;
	int spaceSize = sizeof(space) * numSpaces;

	int numWalls = WALL_LENGTH * WALL_WIDTH;
	int wallSize = sizeof(wall) * numWalls;

	// Malloc the array of wall / board
	wall *walls = (wall *)malloc(wallSize);
	space *board = (space *)malloc(spaceSize);


	// Initialize, zero out the board 
	boardInit(board);

	// Generate walls 
	generateWalls(walls);

	generateBoard(board, walls);


	outputBoard(board);

	free(walls);
	free(board);








	return 0;
}




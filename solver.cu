#include "hip/hip_runtime.h"
/*
	

*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "board.cu"








__global__ void
solveForAllWalls() {
	int tidx = threadIdx.x;	// X-Dim = Wall
	int tidy = threadIdx.y;	// Y-Dim = Direction
	int space = blockIdx.x;	// Space #

	// 


}




/* Kernel

*/
__global__ void
calcBestMove() {
	int gtid, block, tidx, tidy;
	tidx = threadIdx.x;
	tidy = threadIdx.y;
	block = blockIdx.x;

	gtid = block * blockDim.x + tidx + tidy;
	
	// Block = Space
	// x = tid.x 	wall
	// y = tid.y	wall direction 0 - 4

	// moveWall (walls, tid, newDir, position)

	CUDA_moveWallParallel2D(walls,)

}




// CUDA Error Check
void checkCudaError(hipError_t e, char in[]) {
	if (e != hipSuccess) {
		printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
		exit(EXIT_FAILURE);
	}
}


int main(int argc, char const *argv[])
{
	
	int numSpaces = SPACE_LENGTH * SPACE_WIDTH;
	size_t spaceSize = sizeof(space) * numSpaces;

	int numWalls = WALL_LENGTH * WALL_WIDTH;
	size_t wallSize = sizeof(wall) * numWalls;

	// Malloc the array of wall / board
	wall *walls = (wall *)malloc(wallSize);
	space *board = (space *)malloc(spaceSize);


	// Initialize, zero out the board 
	boardInit(board);
	// Generate walls 
	generateWalls(walls);
	generateBoard(board, walls);



	// Malloc space on device, copy to device
	walls *d_walls = NULL;
	space *d_board = NULL;

	checkCudaError( hipMalloc((void**) &d_walls, wallSize), 
		"Malloc Histogram");
	checkCudaError( hipMalloc((void**) &d_board, spaceSize), 
		"Malloc Atom List");

	checkCudaError( hipMemcpy(d_histogram, histogram, histogramSize, hipMemcpyHostToDevice), 
		"Copy histogram to Device");
	checkCudaError( hipMemcpy(d_atom_list, atom_list, atomSize, hipMemcpyHostToDevice), 
		"Copy atom_list to Device");




	// Setup: Measure Runtime
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// CUDA Kernel Call
	//PDH_baseline <<<ceil(PDH_acnt/32), 32>>> (d_histogram, d_atom_list, PDH_res, PDH_acnt);

	checkCudaError(hipGetLastError(), "Checking Last Error, Kernel Launch");

	// Report kernel runtime
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate: %f ms \n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);




	return 0;
}